#include "hip/hip_runtime.h"
#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "Test.cu"
#include "Constants.cu"
#include "Init.cu"
#include "Errors.cu"
#include "PrintOnScreen.cu"
#include "GetSetsFromFile.cu"
#include "GetDefiniedItems.cu"
#include "BoolToIntConvert.cu"
#include "MakeBitmap.cu"
#include "GetRowSupremum.cu"
#include "GetFrequentSets.cu"
#include "GetRowTableBitAnd.cu"
#include "GetTableToBitAnd.cu"
#include "FindSets.cu"

#include "HostGetRowTableBitAnd.cu"
#include "HostFindSets.cu"

const int NUMBER_OF_ITERATIONS = 1;

unsigned int* gAndRow;
unsigned int** gFrequentSets;
unsigned int** gAndTable;
int* gFrequentSetsCount;
unsigned int* gIntVector;
bool* gBoolPacket;
bool** gInputBitmap;
unsigned int** gIntTable;
unsigned int** gFrequentBitmap;
int* gNewBitmapRowCount;
unsigned int** gNewBitmap;
unsigned int* gTable1D;
unsigned int** gAndTable2;

unsigned int testCount1;
unsigned int testCount2;

/*__device__*/ unsigned int *devRow1, *devRow2;
/*__device__*/ unsigned int* devGlobalAndTable;
/*__device__*/ unsigned int* devRow;

void StartCpuAndGpuFindSets()
{
	int** setsFromFile = GetSetsFromFile();
	int* definiedItems = GetDefinedItems();
	unsigned int** bitmap = MakeBitmap(setsFromFile, definiedItems);
	int* frequentSetsCount = (int*)malloc(sizeof(int*));
	unsigned int** frequentBitmap = GetFrequentSets(bitmap,
											NUMBER_OF_DIFFERENT_ITEMS,
											frequentSetsCount);

	clock_t startCPU, endCPU, startGPU, endGPU;
testCount1 = 0;
testCount2 = 0;

	startCPU = clock();
	__int64 startCpuTicks = GetTicks();

printf("this is start of cpu\n");
	//int i;
	//for(i = 0; i < NUMBER_OF_ITERATIONS; ++i)
	//{
		FindSets(frequentBitmap, *frequentSetsCount);
	//}
		printf("testCount: %d\n",testCount1);
printf("this is end of cpu\n");

	__int64 endCpuTicks = GetTicks();
	endCPU = clock();

	//printf("->End of CPU, start of GPU\n");

	startGPU = clock();
	__int64 startGpuTicks = GetTicks();
printf("this is start of GPU\n");
	//for(i = 0; i < NUMBER_OF_ITERATIONS; ++i)
	//{
		HostFindSets(frequentBitmap, *frequentSetsCount);
	//}
				printf("testCount: %d\n",testCount2);
printf("this is end of GPU\n");

	__int64 endGpuTicks = GetTicks();
	endGPU = clock();

	double cpuTime = (double)( endCPU - startCPU ) / (double)CLOCKS_PER_SEC;
	double gpuTime = (double)( endGPU - startGPU ) / (double)CLOCKS_PER_SEC;

	printf ("CPU processing time: %f [s]\n", cpuTime);
	printf ("GPU processing time: %f [s]\n", gpuTime);

	if(cpuTime < gpuTime)
	{
		double ratio = gpuTime / cpuTime;
		printf("\nCPU was faster %f times\n", ratio);
	}
	else
	{
		double ratio = cpuTime / gpuTime;
		printf("\nCUDA SUSSESS! GPU was faster %f times\n", ratio);
	}

	printf ("\nTicks summary:\n");
	printf ("Cpu ticks: ");
	PrintTicks(startCpuTicks, endCpuTicks);
	printf ("\nGpu ticks: ");
	PrintTicks(startGpuTicks, endGpuTicks);
}

//------------------------------ Start of HostGetRowTableBitAnd.cu file ---------------------------------//
// Kernel that executes on the CUDA device
__global__ void CudaGetBitAndOfRows(unsigned int* table1D, unsigned int* row, int rowSize, int tableRowCount)
{
	  int idx = blockIdx.x * blockDim.x + threadIdx.x;

	  if (idx < tableRowCount * rowSize)
	  {
		table1D[idx] = table1D[idx] & row[idx % rowSize];
	  }
}

void RunCudaGetBitAndOfRows(unsigned int* row, unsigned int* table1D, int tableRowCount)
{
	//__int64 start1 = GetTicks();
	//hipMemcpy(devRow1, row1, NUMBER_OF_INTS_IN_ROW, hipMemcpyHostToDevice);
	//hipMemcpy(devRow2, row2, NUMBER_OF_INTS_IN_ROW, hipMemcpyHostToDevice);
	hipMemcpy(devGlobalAndTable,
			   table1D,
			   tableRowCount * NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int),
			   hipMemcpyHostToDevice);

	hipMemcpy(devRow, row, sizeof(unsigned int*), hipMemcpyHostToDevice);
//__int64 stop1 = GetTicks();

	int blockSize = 32;
	int nBlocks = NUMBER_OF_INTS_IN_ROW / blockSize + (NUMBER_OF_INTS_IN_ROW % blockSize == 0 ? 0:1);
//__int64 start3 = GetTicks();
	CudaGetBitAndOfRows <<< nBlocks, blockSize >>> (devGlobalAndTable,
													devRow,
													NUMBER_OF_INTS_IN_ROW,
													tableRowCount);
//__int64 stop3 = GetTicks();
//__int64 start4 = GetTicks();
	hipMemcpy(table1D,
			   devGlobalAndTable,
			   tableRowCount * NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int),
			   hipMemcpyDeviceToHost);
//__int64 stop4 = GetTicks();

	/*printf ("\n\nRunCudaGetBitAndOfRoww total: ");
	PrintTicks(start1 , stop4);
	printf ("\ncudaMemcpy1: ");
	PrintTicks(start1 , stop1);
	printf ("\nCudaGetBitAndOfRows: ");
	PrintTicks(start3 , stop3);
	printf ("\ncudaMemcpy3: ");
	PrintTicks(start4 , stop4);*/
}

unsigned int** HostGetRowTableBitAnd(unsigned int* row, unsigned int** table, int tableRowCount)
{
	unsigned int* table1D =
		(unsigned int*)malloc(tableRowCount * NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));

	//__int64 start1 = GetTicks();

	int i,j;
	for(i = 0; i < tableRowCount; ++i)
	{
		for(j = 0; j < NUMBER_OF_INTS_IN_ROW; ++j)
		{
			table1D[i * NUMBER_OF_INTS_IN_ROW + j ] = table[i][j];
		}
	}
	//__int64 stop1 = GetTicks();

	//__int64 start2 = GetTicks();
	RunCudaGetBitAndOfRows(row, table1D, tableRowCount);
	//__int64 stop2 = GetTicks();

	unsigned int** andTable = (unsigned int**)malloc(tableRowCount * sizeof(unsigned int*));
	//__int64 start3 = GetTicks();
	for(i = 0; i < tableRowCount; ++i)
	{
		andTable[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(int));
		for(j = 0; j < NUMBER_OF_INTS_IN_ROW; ++j)
		{
			andTable[i][j] = table1D[i * NUMBER_OF_INTS_IN_ROW + j ];
		}
	}
	//__int64 stop3 = GetTicks();

/*	printf ("\n\nGpu statistics: ");
	printf ("Total time: ");
	PrintTicks(start1 , stop3);
	printf ("\nFirst loop: ");
	PrintTicks(start1 , stop1);
	printf ("\nBit and on gpu: ");
	PrintTicks(start2 , stop2);
	printf ("\nSecond loop: ");
	PrintTicks(start3 , stop3);*/
/*
	unsigned int** andTable = (unsigned int**)malloc(tableRowCount * sizeof(unsigned int*));
	int i,j;
	for(i = 0; i < tableRowCount; ++i)
	{
		andTable[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));
		*andTable[i] = *row;
		RunCudaGetBitAndOfRows(andTable[i], table[i]);
	}
*/
	

	return andTable;
}

//------------------ Test ------------------//
void TestRunCudaGetBitAndOfRows()
{
	PrintTestStartMessage("RunCudaGetBitAndOfRows");

	int** setsFromFile = GetSetsFromFile();
	int* definiedItems = GetDefinedItems();
	unsigned int** bitmap = MakeBitmap(setsFromFile, definiedItems);

//	RunCudaGetBitAndOfRows(bitmap[0], bitmap[1]);

	PrintTab();

	int i;
	for(i = 0; i < NUMBER_OF_INTS_IN_ROW; ++i)
		printf("%d ", bitmap[0][i]);

	PrintNL();

	PrintTestPassedMessage("RunCudaGetBitAndOfRows");
}

void TestHostGetRowTableBitAnd()
{
	PrintTestStartMessage("HostGetRowTableBitAnd");

	int** setsFromFile = GetSetsFromFile();
	int* definiedItems = GetDefinedItems();
	unsigned int** bitmap = MakeBitmap(setsFromFile, definiedItems);

	unsigned int** bitAndTable = HostGetRowTableBitAnd(bitmap[0],
											   bitmap,
											   NUMBER_OF_DIFFERENT_ITEMS);
	int i,j;
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
	{
		PrintTab();

		for(j = 0; j < NUMBER_OF_INTS_IN_ROW; ++j)
			printf("%d ", bitAndTable[i][j]);

		PrintNL();
	}

	PrintTestPassedMessage("HostGetRowTableBitAnd");
}

//------------------ End of Test ------------------//

//------------------------------ End of HostGetRowTableBitAnd.cu file ---------------------------------//

int main(void)
{
	InitVariables(); //This function must be run before other operations can begin
					 //Don't remove or comment this function!

	int i;

	gAndTable2 = (unsigned int**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(unsigned int*));
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
		gAndTable2[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(int));

	gTable1D =
		(unsigned int*)malloc(NUMBER_OF_DIFFERENT_ITEMS * NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));

	gNewBitmap = (unsigned int**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(unsigned int*));
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
		gNewBitmap[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));

	gNewBitmapRowCount = (int*)malloc(sizeof(int*));

/* Global pointers memory allocation */
    gAndRow = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));
	gFrequentSets = (unsigned int**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(unsigned int*) );
	
	gAndTable = (unsigned int**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(unsigned int*));
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
		gAndTable[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));
	
	gFrequentSetsCount = (int*)malloc(sizeof(int*));
	gIntVector = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int));
	gBoolPacket = (bool*)malloc(NUMBER_OF_BITS_IN_INT * sizeof(bool));
	
	gInputBitmap = (bool**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(bool*));
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
		gInputBitmap[i] = (bool*)malloc(NUMBER_OF_TRANSACTIONS * sizeof(bool));
	
	gIntTable = (unsigned int**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(unsigned int*));
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
		gIntTable[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(int*));
	
	gFrequentBitmap = (unsigned int**)malloc(NUMBER_OF_DIFFERENT_ITEMS * sizeof(unsigned int*));;
	for(i = 0; i < NUMBER_OF_DIFFERENT_ITEMS; ++i)
		gFrequentBitmap[i] = (unsigned int*)malloc(NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int*));

/* End of global pointers memory allocation */

	/*Don't remove or comment two lines belove. It is memeory allocation for GPU variables.*/
	//hipMalloc((void**) &devRow1, NUMBER_OF_INTS_IN_ROW);
	//hipMalloc((void**) &devRow2, NUMBER_OF_INTS_IN_ROW);

	hipMalloc((void**) &devGlobalAndTable,
			   ((NUMBER_OF_DIFFERENT_ITEMS - 1) * NUMBER_OF_INTS_IN_ROW) * sizeof(unsigned int) );
	hipMalloc((void**) &devRow, NUMBER_OF_INTS_IN_ROW * sizeof(unsigned int) );

	//CPU testing
	//TestGetSetsFromFile();
	//TestMakeBitmapBool();
	//TestMakeIntVectorFromBitVector();
	/*TestMakeBitmap();
	TestGetTableToBitAnd();
	TestGetRowTableBitAnd();
	TestFindSets();*/

	//GPU testting
	//TestRunCudaGetBitAndOfRows();
	//TestHostFindSets();

	StartCpuAndGpuFindSets();

/* Free global pointers memory*/
    free(gAndRow);
	free(gFrequentSets);
	free(gAndTable);
	free(gFrequentSetsCount);
	free(gIntVector);
	free(gBoolPacket);
//	for(i = 0; i < NUMBER_OF_TRANSACTIONS; ++i)
//		free(gInputBitmap[i]);
	free(gInputBitmap);
	free(gIntTable);
/* End of free global pointers memory */

	//hipFree(&devRow1);
	//hipFree(&devRow2);
	hipFree(&devGlobalAndTable);
	hipFree(&devRow);

	getchar();
}
