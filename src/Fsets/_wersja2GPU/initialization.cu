#include "hip/hip_runtime.h"
#include "initialization.h"






void generateData(int* tab) {
	int items[frequentItemsetSize];
	memset(tab,0,dataTableSizeInBytes); //wyczysc tabele z danymi
	
	for (int i=0;i<frequentItemsetsToGenerate;i++) {
		for (int j=0;j<frequentItemsetSize;j++) {
			items[j]=(int)round(((distinctItemCount-1)*getDoubleRand()));
		}
		for (int j=0;j<1.1*minsup;j++) {
			int col=(int)round((itemsetCount-1)*getDoubleRand());
			for (int k=0;k<frequentItemsetSize;k++) {
				tab[items[k]*rowSizeIntMultiple+col/32]|=1<<col%32;
			}
		}
	}

}

//Zadziała poprawnie jedynie dla distinctItemCount=3 i itemsetCount=256. minsup=4
void generateData2(int* tab) {
	int rob[]={
		1,1,1,1,0,1,1,0,
		0,0,1,1,1,0,1,1,
		1,1,1,1,1,1,1,0};
	memcpy(tab,rob,24*4);
}

void generateData3(int* tab) {
	int rob[]={
		1,1,1,1,0,0,0,0,
		0,0,0,0,1,1,1,1,
		1,1,1,1,0,0,0,0,
		0,0,0,0,1,1,1,1,
		1,1,1,1,0,0,0,0};
	memcpy(tab,rob,5*8*sizeof(int));
}