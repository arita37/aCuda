#include <math.h>
#include <stdio.h>
#include <cutil_inline.h>
#include "cudpp.h"
#include "itemsetsearch.h"

void displayDataTableGPU(int* d_dataToDisplay,int cols,int rows,char* format,char* title) {
	int* buffer=(int*)malloc(cols*rows*sizeof(int));
	cutilSafeCall(hipMemcpy(buffer,d_dataToDisplay,cols*rows*sizeof(int),hipMemcpyDeviceToHost));
	displayDataTableCPU(buffer,cols,rows,format,title);
	free(buffer);
}

void allocDeviceMemoryForFilter() {
	cutilSafeCall(hipMalloc((void**) &d_data, dataTableSizeInBytes));
	cutilSafeCall(hipMalloc((void**) &d_countOnes, dataTableSizeInBytes));

	int threads=f2plog2int(rowSizeIntMultiple);
	threads=threads>512?512:threads;
	int blocks=rowSizeIntMultiple/threads;


	countTempRowIntMultiple=blocks; 
	countTempSizeInBytes=countTempRowIntMultiple*sizeof(int)*distinctItemCount;

	cutilSafeCall(hipMalloc((void**) &d_countTemp, countTempSizeInBytes)); 
	cutilSafeCall(hipMalloc((void**) &d_supportScan,sizeof(int)*(distinctItemCount+1)));
	cutilSafeCall(hipMalloc((void**) &d_supportIndexes,sizeof(int)*distinctItemCount));
	cutilSafeCall(hipMalloc((void**) &d_temp, 512*sizeof(int)));
	cutilSafeCall(hipMalloc((void**) &d_reducedData, dataTableSizeInBytes));
	
	
	DEBUGONLY(printf("Device memory for filter allocated\n");)
}

void freeDeviceMemoryForFilter() {
	//cutilSafeCall(hipFree(d_supportScan));
	//cutilSafeCall(hipFree(d_supportIndexes));
	//cutilSafeCall(hipFree(d_temp));
	//cutilSafeCall(hipFree(d_rowNumbers));
	cutilSafeCall(hipFree(d_countOnes));
	cutilSafeCall(hipFree(d_countTemp));
	cutilSafeCall(hipFree(d_data));
	DEBUGONLY(printf("Device memory for filter freed\n");)
}

void allocDeviceMemoryForAlgo(int rows) {
	//cutilSafeCall(hipMalloc((void**) &d_data, rows*rowSizeIntMultiple*sizeof(int)));
	d_data=d_reducedData;
	
	cutilSafeCall(hipMalloc((void**) &d_tempData, rows*rowSizeIntMultiple*sizeof(int)));
	cutilSafeCall(hipMalloc((void**) &d_countOnes,  rows*rowSizeIntMultiple*sizeof(int)));


	int threads=f2plog2int(rowSizeIntMultiple);
	threads=threads>512?512:threads;
	int blocks=rowSizeIntMultiple/threads;


	countTempRowIntMultiple=blocks; 
	countTempSizeInBytes=countTempRowIntMultiple*sizeof(int)*rows;

	cutilSafeCall(hipMalloc((void**) &d_countTemp, countTempSizeInBytes)); 


	d_levelsData=(int**)malloc(maxRecursion*sizeof(int*));
	d_levelsIndexes=(int**)malloc(maxRecursion*sizeof(int*));
	d_levelsRowNumbers=(int**)malloc(maxRecursion*sizeof(int*));
	

	for (int i=0;i<maxRecursion;i++) {
		cutilSafeCall(hipMalloc((void**) &d_levelsData[i],rows*rowSizeIntMultiple*sizeof(int)));
		cutilSafeCall(hipMalloc((void**) &d_levelsIndexes[i],rows*sizeof(int)));
		cutilSafeCall(hipMalloc((void**) &d_levelsRowNumbers[i], distinctItemCount*sizeof(int)));
	}

	DEBUGONLY(printf("Device memory for algo allocated\n");)
}

void freeDeviceMemoryForAlgo(int rows) {
	cutilSafeCall(hipFree( d_reducedData));
	cutilSafeCall(hipFree( d_tempData));
	cutilSafeCall(hipFree(d_supportIndexes));
	
	for (int i=0;i<maxRecursion;i++) {	
		cutilSafeCall(hipFree(d_levelsData[i]));
		cutilSafeCall(hipFree(d_levelsIndexes[i]));
		cutilSafeCall(hipFree(d_levelsRowNumbers[i]));
	}
	free(d_levelsData);
	free(d_levelsIndexes);
	free(d_levelsRowNumbers);
	
	DEBUGONLY(printf("Device memory for algo freed\n");)
}

int main(int argc, char** argv) {
	//printf("%d\n",RAND_MAX);
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

	//runBinarySingleBlock();
	//runHostBinary();
	//runBinarySingleBlock();
	/*runSearchHost();
	runSearchDevice();
	runSearchDevice2();*/
	runSearchHost();
	//playWithScan();
	//int a;
	//scanf("%d",&a);
	hipDeviceReset();
	getchar();
}

