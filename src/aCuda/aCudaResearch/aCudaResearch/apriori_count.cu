
#include <hip/hip_runtime.h>
#include <iostream>

extern "C" __global__ void count_frequency(int * input, int * output, unsigned width, unsigned height) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int elementIndex = baseX; elementIndex < width; elementIndex += totalThreads) 
	{
		int i = elementIndex;
		int sum = 0;

		while(i < width * height) 
		{
			sum += input[i];
			i += width;
		}

		output[elementIndex] = sum;
	}
}

extern "C" __global__ void count_sets_frequencies(int * input, int * inputSets, int * output, 
												 unsigned width, unsigned height, unsigned setWidth, unsigned sets, unsigned minSup) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int cid = baseX; cid < sets; cid += totalThreads) 
	{
		int occuredSum = 0;
		int startPoint = cid * setWidth;

		for(int tid = 0; tid < width; tid++) 
		{
			bool yes = true;
			int i = 0;

			while(i < setWidth) 
			{
				if (input[inputSets[i + startPoint] * width + tid] != 1)
				{
					yes = false;
					i = setWidth;
				} 
				else 
				{
					i++;	
				}
			}

			// we found in the transaction all elements from the checked set
			if(yes) 
			{
				occuredSum++;	
			}

			if(occuredSum > minSup)
					break;
		}

		output[cid] = occuredSum;
	}
}
/*
extern "C" __global__ void count_frequency_matrix(int * input, int * inputSets, int * output, 
												 unsigned width, unsigned height, unsigned setWidth, unsigned sets) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int c = 0; c < sets; c++) 
	{
		int occuredSum = 0;

		for(int tid = baseX; tid < height; tid += totalThreads) 
		{
			bool yes = true;

			for(int i = 0; i < setWidth; i++)
			{
				if (input[tid * width + inputSets[i + c * setWidth]] != 1)
				{
					yes = false;
					break;
				}
			}
			// we found in the transaction all elements from the checked set
			if(yes) 
			{
				occuredSum++;	
			}
		}

		output[baseX + c * totalThreads] = occuredSum;
	}
}

extern "C" __global__ void count_frequency_matrix2(int * input, int * inputSets, int * output, 
												 unsigned width, unsigned height, unsigned setWidth, unsigned sets) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;


		for(int tid = baseX; tid < height; tid += totalThreads) 
		{	
			for(int c = 0; c < sets; c++) 
			{
				int occuredSum = output[baseX + c * totalThreads];

				bool yes = true;

				for(int i = 0; i < setWidth; i++)
				{
					if (input[tid * width + inputSets[i + c * setWidth]] != 1)
					{
						yes = false;
						break;
					}
				}
				// we found in the transaction all elements from the checked set
				if(yes) 
				{
					occuredSum++;	
					output[baseX + c * totalThreads] = occuredSum;
				}
			}
		}

}*/

// count sum for each column (which is support for candidate!)
extern "C" __global__ void count_frequency_table(int * input, int * output, 
												 unsigned width, unsigned height) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int id = baseX; id < height; id += totalThreads) 
	{
		int sum = 0;
		int innerId = 0;

		while(innerId < width) 
		{
			sum += input[innerId + id * width];
			innerId += 1;
		}

		output[id] = sum;
	}
}