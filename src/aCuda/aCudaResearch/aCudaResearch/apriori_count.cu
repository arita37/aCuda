
#include <hip/hip_runtime.h>
#include <iostream>

/*extern "C" __global__ void add(int a, int b, int *c) {
	*c = a + b;
}*/

extern "C" __global__ void count_frequency(int * input, int * output, unsigned width, unsigned height) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int elementIndex = baseX; elementIndex < width; elementIndex += totalThreads) {

		int i = elementIndex;
		int sum = 0;

		while(i < width * height) 
		{
			sum += input[i];
			i += width;
		}

		output[elementIndex] = sum;
	}
}
/*
extern "C" __global__ void add_vector(int *a, int *b, int *c) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	while(tid < 10) {
		c[tid] = a[tid] + b[tid];
		tid += totalThreads;
	}
}*/