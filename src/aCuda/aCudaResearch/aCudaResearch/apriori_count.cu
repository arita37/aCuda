
#include <hip/hip_runtime.h>
#include <iostream>

extern "C" __global__ void count_frequency(int * input, int * output, unsigned width, unsigned height) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int elementIndex = baseX; elementIndex < width; elementIndex += totalThreads) 
	{
		int i = elementIndex;
		int sum = 0;

		while(i < width * height) 
		{
			sum += input[i];
			i += width;
		}

		output[elementIndex] = sum;
	}
}

extern "C" __global__ void count_frequency_matrix(int * input, int * inputSets, int * output, 
												 unsigned width, unsigned height, unsigned setWidth, unsigned sets) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int c = 0; c < sets; c++) 
	{
		int occuredSum = 0;

		for(int tid = baseX; tid < height; tid += totalThreads) 
		{
			int sum = 0;
			for(int i = 0; i < setWidth; i++)
			{
				sum += input[tid * width + inputSets[i]];
			}
			// we found in the transaction all elements from the checked set
			if(sum == setWidth) 
			{
				occuredSum++;	
			}
		}

		output[baseX + c * totalThreads] = occuredSum;
	}
}

// count sum for each column (which is support for candidate!)
extern "C" __global__ void count_frequency_table(int * input, int * output, 
												 unsigned width, unsigned height) 
{
	int baseX = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for(int id = baseX; id < width; id += totalThreads) 
	{
		int sum = 0;

		while(id < width * height) 
		{
			sum += input[id];
			id += width;
		}

		output[id] = input[id];
	}
}